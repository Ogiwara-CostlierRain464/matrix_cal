#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

__global__ void matMul(const half* const a_ptr,
                       const half* const b_ptr,
                       half* const c_ptr){
    // cの一行目はbの1行…となるように計算
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t _i = 0; _i < 1'000'000'0; _i++){
        for(size_t i = 0; i < 16; i++){
            c_ptr[i * 16 + idx] = b_ptr[i * 16 + idx] + half(4);
        }
    }
}

int main(int argc, char** argv){
    half *a;
    hipMalloc((void**)  &a, 16 * 16 );
    half *b;
    hipMalloc((void**)  &b, 16 * 16 );
    half *c;
    hipMalloc((void**)  &c, 16 * 16 );

    dim3 grid(4);
    dim3 block(4); // 16スレッド並列

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matMul<<<grid, block>>>(a, b, c);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time: " << milliseconds << " ms" << std::endl;

    return 0;
}