#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <functional>
#include <array>
#include <cassert>
#include <set>


#define M 16
#define K 16
#define N 32768 * 8
#define ITER_NUM 1000
#define THREAD_BLOCK_SIZE 32
#define W_MAP_WIDTH K / 4

// for H100 256K bytes
#define SHARED_MEM_SIZE 256000

#define BEGIN_ITER for(size_t i = 0; i < ITER_NUM; i++){
#define END_ITER   }

__device__ signed char W_mat[M * K]; // row major
// TODO X map should support dynamic length
// I just fill this matrix with index num
__device__ unsigned short W_map[W_MAP_WIDTH * M]; // row major



/**
 * Prepare both W_mat and W_map before the measurement.
 */
__global__ void prepareW(){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= M){
        // this thread won't work for init
        return;
    }

    int row = tid;

    for(size_t col = 0; col < (K / 4); col++){
        W_map[row * (K / 4) + col] = col;
    }

    for(size_t col = 0; col < K; col++){
        if(col < (K / 4)){
            W_mat[row * K + col] = 1;
        }else{
            W_mat[row * K + col] = 0;
        }
    }
}



__global__ void tcMatMul(const signed char* const X,
                       int* const c){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> W_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> X_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> c_frag;

    // thread blockあたり用意してあげる
    //__shared__ signed char[];

    BEGIN_ITER

    nvcuda::wmma::fill_fragment(c_frag, 0);

    for(size_t k = 0; k < K; k += 16){
        nvcuda::wmma::load_matrix_sync(W_frag, W_mat + (blockIdx.y * K * 16 + k), K);
        nvcuda::wmma::load_matrix_sync(X_frag, X + ( k * N + blockIdx.x * 16) , N);
        nvcuda::wmma::mma_sync(c_frag, W_frag, X_frag, c_frag);
    }

    nvcuda::wmma::store_matrix_sync(c + (blockIdx.y * N * 16 + blockIdx.x * 16), c_frag, N, nvcuda::wmma::mem_row_major);

    END_ITER
}

__device__ char (* copyToShared(const char* const X_g))[THREAD_BLOCK_SIZE]
{
    /**
     * A100のshared memoryは164KB
     * H100のshared memoryは256KB
     * 0番目のスレッドがアクセスするのは、X行列の0列
     *
     */
     __shared__ char X_s[K][THREAD_BLOCK_SIZE];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t row = 0; row < K; row++){
        for(size_t col = 0; col < THREAD_BLOCK_SIZE; col++){
            X_s[row][col] = X_g[row * N + tid];
        }
    }
    return X_s;
}

__global__ void cuMatMul(const char* const X, int* const c){


    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    char (*X_s)[THREAD_BLOCK_SIZE] = copyToShared(X);
    __shared__ short W_map_s[M][W_MAP_WIDTH];

    for(size_t row = 0; row < M; row++){
        for(size_t col = 0; col < W_MAP_WIDTH; col++){
            // Just copy
            W_map_s[row][col] = W_map[row * W_MAP_WIDTH + col];
        }
    }

    __shared__ char c_s[M][THREAD_BLOCK_SIZE];

    static_assert(K * THREAD_BLOCK_SIZE + M * W_MAP_WIDTH + M * THREAD_BLOCK_SIZE <= SHARED_MEM_SIZE);

    BEGIN_ITER

    for(size_t row = 0; row < M; row++){
        int accum = 0;
        for(size_t i = 0; i < (K/4); i++){
            accum += X_s[W_map_s[row][i]][local_tid];
        }
        c_s[row][local_tid] = accum;
        //c[row * N + col] = accum;

        /**
         * col = 0, row = 1の時: c[1 * N + 0] => c[N] , c[2N], c[3N], c[4N] …と、飛び飛び？　
         * col = 1, row = 1の時: c[1 * N + 1] => c[N+1], c[2N+1], …と、飛び飛び？
         * だが、メモリの性質はうまく利用している？
         */
    }

    END_ITER

    for(size_t row = 0; row < M; row++){
        c[row * N + tid] = c_s[row][local_tid];
    }

}

// cをcolumn orderで管理する
__global__ void cuMatMulCol(const char* const X, int *c){

    BEGIN_ITER

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t row = 0; row < M; row++){
        int accum = 0;
        for(size_t i = 0; i < (K/4); i++){
            accum += X[ W_map[row * (K/4) + i] * N + col ];
        }
        c[col * (M+1) + row] = accum;
        /**
         * col = 0, row = 0, 1, 2, 3の時: c[0 * (M+1) + 0] => c[0] , c[1], c[2], c[3] …と隣接
         * col = 1, row = 0, 1, 2, 3の時: c[1 * (M+1) + 0] => c[M+1], c[M+2], c[M+3], c[M+4]…と隣接
         * col = 2, row = 0, 1, 2, 3の時: c[2 * (M+1) + 0] => c[2M+2], c[2M+3], c[2M+4], c[2M+5]…と隣接
         *
         * TODO: shared memoryに移動
         *
         * https://toropippi.livedoor.blog/archives/55467682.html
         */
    }

    END_ITER
}

float measureKernel(std::function<void(void)> fn){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    fn();

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

void make_J(std::array<char, K * N> *X){
    X->fill(1);
}

void make_I(std::array<char, K * N> *X){
    for(size_t row = 0; row < K; row++){
        for(size_t col = 0; col < N; col++){
            if(row == col) {
                X->at(row * N + col) = 1;
            }
        }
    }
}

/**
 * Calc matmul of W(MxK) and X(KxN), where W is ternary matrix and X is 8-bit matrix.
 * Since X is 8-bit, we need to implement W as a 8-bit matrix due to restriction of wmma.
 * W is prepared before the performance measure.
 */
int main(int argc, char** argv){

    static_assert(M % 16 == 0 && "mod 16 should be 0");
    static_assert(K % 16 == 0 && "mod 16 should be 0");
    static_assert(N % 16 == 0 && "mod 16 should be 0");
    static_assert(K < 65536 && "K should be fit in the maximum of unsigned short");

    // row major
    char *X_d;
    hipMalloc((void**)  &X_d, sizeof(char) * K * N );
    auto *X_ar = new std::array<char, K * N>(); make_J(X_ar);
    hipMemcpy(X_d, X_ar->data(), K * N * sizeof(char), hipMemcpyHostToDevice);

    int *c_d; hipMalloc((void**)  &c_d, sizeof(int) * M * N ); hipMemset(c_d, 0, sizeof(int) * M * N);
    auto c_ar = new std::array<int, N * 1>(); // store only first row

    int *c_with_bank_d; hipMalloc((void**)  &c_with_bank_d, sizeof(int) * N * (M+1) ); hipMemset(c_with_bank_d, 0, sizeof(int) * N * (M+1));

    prepareW<<< M / 16, 16>>>();
    hipDeviceSynchronize();

    std::cout << "Start: " << "M=" << M << " K=" << K << " N=" << N << " ITER=" << ITER_NUM << std::endl;

    float ms = measureKernel([X_d, c_d](){
        tcMatMul<<< dim3(N / 16, M / 16) , 32>>>(( signed char * )  X_d, c_d);
    });
    std::cout << "TensorCore Time: " << ms << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == K / 4 && "what");

    ms = measureKernel([X_d, c_d](){
        cuMatMul<<<(N / 32) , 32>>>(X_d, c_d);
    });
    std::cout << "CudaCore Time: " << ms << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == K / 4 && "what");

    ms = measureKernel([X_d, c_with_bank_d](){
        cuMatMulCol<<<(N / 32) , 32>>>(X_d, c_with_bank_d);
    });
    std::cout << "CU Column Time: " << ms << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_with_bank_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == K / 4 && "what");

    return 0;
}