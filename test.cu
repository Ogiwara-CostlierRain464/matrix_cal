#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <functional>
#include <array>
#include <cassert>
#include <set>

__global__ void tcMatMul(const signed char* const a,
                       const signed char* const b,
                       int* const c){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::col_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::col_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> c_frag;

    nvcuda::wmma::load_matrix_sync(a_frag, a, 16);
    nvcuda::wmma::load_matrix_sync(b_frag, b, 16);
    nvcuda::wmma::fill_fragment(c_frag, 0);

    nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    nvcuda::wmma::store_matrix_sync(c, c_frag, 16, nvcuda::wmma::mem_col_major);
}

__global__ void cuMatMul(const char* const a,
                       const char* const b,
                       int* const c){
    // assert elements in a must be {-1, 0, 1}

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t row = 0; row < 16; row++){
        int accum = 0;
        for(size_t i = 0; i < 16; i++){
            if(a[row * 16 + i] == 1) {
                accum += b[i * 16 + col];
            }
        }
        c[row * 16 + col] = accum;
    }
}

float measureKernel(std::function<void(void)> fn){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    fn();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

void make_matrix_from_arr(std::array<std::set<size_t>, 16> &arr, std::array<char, 256> &a){
    a.fill(0);

    for(size_t row = 0; row < 16; row++){
        for(auto &e : arr[row]){
            a[row * 16 + e] = 1;
        }
    }
}

void make_binary(std::array<char, 256> &a){
    a.fill(0);

    // 1+1が右端と左端にある
    for(size_t r = 0; r < 16; r++){
        a[0 + r * 16] = 1;
        a[1 + r * 16] = 1;
        a[14 + r * 16] = 1;
        a[15 + r * 16] = 1;
    }
}

void make_I(std::array<char, 256> &b){
    b.fill(0);
    for(size_t i = 0; i < 16; i++){
        b.at(i + i * 16) = 1;
    }
}

int main(int argc, char** argv){
    std::array<std::set<size_t>, 16> sparse;
    for(size_t i = 0; i < 16; i++){
        sparse[i] = {0, 1, 14, 15};
    }
    std::array<char, 256> sparse_matrix;
    make_matrix_from_arr(sparse, sparse_matrix);

    char *a_d; hipMalloc((void**)  &a_d, sizeof(char) * 16 * 16 );
    hipMemcpy(a_d, sparse_matrix.data(), 256 * sizeof(char), hipMemcpyHostToDevice);


    char *b_d; hipMalloc((void**)  &b_d, sizeof(char) * 16 * 16 );
    std::array<char, 256> b_ar; make_I(b_ar);
    hipMemcpy(b_d, b_ar.data(), 256 * sizeof(char), hipMemcpyHostToDevice);

    int *c_d; hipMalloc((void**)  &c_d, sizeof(int) * 16 * 16 ); hipMemset(c_d, 0, sizeof(int) * 16 * 16);
    std::array<int, 256> c_ar;

    float ms = measureKernel([a_d, b_d, c_d](){
        // 32でないとだめ
        for(size_t i = 0; i < 1000; i++){
            tcMatMul<<<1, 32>>>(( signed char * ) a_d, ( signed char * )  b_d, c_d);
        }
    });
    std::cout << "TensorCore Time: " << ms << "ms" << std::endl;

    hipMemcpy(c_ar.data(), c_d, 256 * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar.at(0) == 1 && "what");

    ms = measureKernel([a_d, b_d, c_d](){
        for(size_t i = 0; i < 1000; i++){
            cuMatMul<<<1, 16>>>( a_d, b_d, c_d);
        }
    });
    std::cout << "CudaCore Time: " << ms << "ms" << std::endl;
    hipMemcpy(c_ar.data(), c_d, 256 * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar.at(0) == 1 && "what");
    assert(c_ar.at(17) == 1 && "what");

    return 0;
}