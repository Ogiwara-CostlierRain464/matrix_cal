#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <functional>
#include <array>
#include <cassert>
#include <set>


#define M 12288L
#define K 12288L
#define N (M * 4L)
#define ITER_NUM 1000

#define W_MAP_LENGTH (K / 10)

#define CALC_M_LENGTH (8L)

__device__ signed char W_mat[M * K];
// TODO X map should support dynamic length
// I just fill this matrix with index num
__device__ short W_map[W_MAP_LENGTH * M];
__device__ short W_map_negative[W_MAP_LENGTH * M];

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

/**
 * Prepare both W_mat and W_map before the measurement.
 */
__global__ void prepareW(){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= M){
        // this thread won't work for init
        return;
    }

    int row = tid;

    for(size_t col = 0; col < W_MAP_LENGTH; col++){
        W_map[row * W_MAP_LENGTH + col] = col;
    }
    for(size_t col = 0; col < W_MAP_LENGTH; col++){
        W_map_negative[row * W_MAP_LENGTH + col] = col;
    }

    for(size_t col = 0; col < K; col++){
        if(col < W_MAP_LENGTH){
            W_mat[row * K + col] = 1;
        }else if(W_MAP_LENGTH <= col && col < W_MAP_LENGTH * 2){
            W_mat[row * K + col] = -1;
        }else{
            W_mat[row * K + col] = 0;
        }
    }
}

__global__ void tcMatMul(const signed char* const X,
                       int* const c){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, nvcuda::wmma::row_major> W_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, nvcuda::wmma::row_major> X_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> c_frag;

    nvcuda::wmma::fill_fragment(c_frag, 0);

    for(size_t k = 0; k < K; k += 16){
        nvcuda::wmma::load_matrix_sync(W_frag, W_mat + (blockIdx.y * K * 16 + k), K);
        nvcuda::wmma::load_matrix_sync(X_frag, X + ( k * N + blockIdx.x * 16) , N);
        nvcuda::wmma::mma_sync(c_frag, W_frag, X_frag, c_frag);
    }

    nvcuda::wmma::store_matrix_sync(c + (blockIdx.y * N * 16 + blockIdx.x * 16), c_frag, N, nvcuda::wmma::mem_row_major);
}

__device__ __forceinline__ short make_sign(short x){
    return  (2 * (short)(x > 0) - 1);
}

// <<< N * M /  CALC_M_LENGTH / 32, 32  >>>
__global__ void cuMatMul2(const char* const X, int* const c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int start_row = (tid / N) * CALC_M_LENGTH;
    int col = tid % N;

#pragma unroll
    for(size_t row = start_row; row < start_row + CALC_M_LENGTH; row++){
        int accum = 0;
#pragma unroll
        for(size_t i = 0; i < W_MAP_LENGTH; i++){
            accum += X[W_map[row * W_MAP_LENGTH + i]];
        }
        // indexを負の値にする方法では、なぜかパフォーマンスが劣化した
        // このため、別のmapとし作成することにより、パフォーマンスの劣化を抑える。
#pragma unroll
        for(size_t i = 0; i < W_MAP_LENGTH; i++){
            accum += -X[W_map_negative[row * W_MAP_LENGTH + i]];
        }
        c[row * N + col] = accum;
    }
}


float measureKernel(std::function<void(void)> fn){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    fn();

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

void make_J(std::array<char, K * N> *X){
    X->fill(1);
}

void make_I(std::array<char, K * N> *X){
    for(size_t row = 0; row < K; row++){
        for(size_t col = 0; col < N; col++){
            if(row == col) {
                X->at(row * N + col) = 1;
            }
        }
    }
}

/**
 * Calc matmul of W(MxK) and X(KxN), where W is ternary matrix and X is 8-bit matrix.
 * Since X is 8-bit, we need to implement W as a 8-bit matrix due to restriction of wmma.
 * W is prepared before the performance measure.
 */
int main(int argc, char** argv){

    static_assert(M % 16 == 0 && "mod 16 should be 0");
    static_assert(K % 16 == 0 && "mod 16 should be 0");
    static_assert(N % 16 == 0 && "mod 16 should be 0");
    static_assert(K < (65536 / 2) && "K should be fit in the maximum of short");

    char *X_d;
    hipMalloc((void**)  &X_d, sizeof(char) * K * N );
    auto *X_ar = new std::array<char, K * N>(); make_J(X_ar);
    hipMemcpy(X_d, X_ar->data(), K * N * sizeof(char), hipMemcpyHostToDevice);

    int *c_d; hipMalloc((void**)  &c_d, sizeof(int) * M * N ); hipMemset(c_d, 0, sizeof(int) * M * N);
    auto c_ar = new std::array<int, N * 1>(); // store only first row

    prepareW<<< M / 16, 16>>>();
    hipDeviceSynchronize();

    std::cout << "Start: " << "M=" << M << " K=" << K << " N=" << N << " ITER=" << ITER_NUM << " W_MAP_LENGTH=" << W_MAP_LENGTH << " CALC_M_LENGTH=" << CALC_M_LENGTH << std::endl;

    float ms = measureKernel([X_d, c_d](){
        for(size_t i = 0; i < ITER_NUM; i++){
            checkKernelErrors((tcMatMul<<< dim3(N / 16, M / 16) , 32>>>(( signed char * )  X_d, c_d)));
        }
    });
    std::cout << "TensorCore Time: " << ms / ((float) ITER_NUM) << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == 0 && "what");

    ms = measureKernel([X_d, c_d](){
        for(size_t i = 0; i < ITER_NUM; i++){
            checkKernelErrors((cuMatMul2<<< N * M / (CALC_M_LENGTH * 32), 32 >>>(X_d, c_d)));
        }
    });
    std::cout << "CudaCore2 Time: " << ms / ((float) ITER_NUM) << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == 0 && "what");


    return 0;
}