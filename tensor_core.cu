#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>


__global__ void matMul(const half* const a_ptr,
                       const half* const b_ptr,
                       half* const c_ptr){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::col_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> c_frag;

    nvcuda::wmma::fill_fragment(c_frag, __float2half(.0f));

    nvcuda::wmma::load_matrix_sync(a_frag, a_ptr, 16);
    nvcuda::wmma::load_matrix_sync(b_frag, b_ptr, 16);

    nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    nvcuda::wmma::store_matrix_sync(c_ptr, c_frag, 16, nvcuda::wmma::mem_col_major);
}

int main(int argc, char** argv){
    // make 16*16 arr

    half *a;
    hipMalloc((void**)  &a, 16 * 16 );
    half *b;
    hipMalloc((void**)  &b, 16 * 16 );
    half *c;
    hipMalloc((void**)  &c, 16 * 16 );

    dim3 grid(1);
    dim3 block(1);

    matMul<<<grid, block>>>(a, b, c);

    printf("Done.\n");

    return 0;
}