#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <functional>
#include <array>
#include <cassert>
#include <set>
#include <type_traits>

#include "submodule/wmma_extension/include/wmma_extension/wmma_extension.hpp"

//#define RUN_TC
//#define RUN_CUDA
#define RUN_NEW

// X: MxK  W: KxN  C: MxN
#define D_MODEL 4096L
#define BATCH_SIZE 4096L // for real-time inference
#define M BATCH_SIZE
#define K D_MODEL
#define N (D_MODEL * 4)
#define ITER_NUM 10

#define W_MAP_LENGTH (K / 20)

#define CALC_N_LENGTH (8L)

#define MAJOR_ROW 0
#define MAJOR_COL 1
#define X_MAJOR MAJOR_COL
#define W_MAJOR MAJOR_COL
#define C_MAJOR MAJOR_COL

#define CAT(x, y) x ## y

#define BT_0(mat, row_dim, col_dim, row, col) mat[row * col_dim + col]
#define BT_1(mat, row_dim, col_dim, row, col) mat[col * row_dim + row]
#define BT(major) CAT(BT_, major)

__device__ signed char W_mat[K * N];
__device__ unsigned short W_map[W_MAP_LENGTH * N];
__device__ unsigned short W_map_negative[W_MAP_LENGTH * N];

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

/**
 * Prepare both W_mat and W_map before the measurement.
 */
__global__ void prepareW(){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= N){
        // this thread won't work for init
        return;
    }

    int col = tid;

    for(size_t row = 0; row < W_MAP_LENGTH; row++){
        BT(W_MAJOR) (W_map, W_MAP_LENGTH , N, row, col) = row;
    }
    for(size_t row = 0; row < W_MAP_LENGTH; row++){
        BT(W_MAJOR) (W_map_negative ,W_MAP_LENGTH ,N, row, col) = row + W_MAP_LENGTH;
    }

    for(size_t row = 0; row < K; row++){
        if(row < W_MAP_LENGTH){
            BT(W_MAJOR) (W_mat, K, N, row, col) = 1;
        }else if(W_MAP_LENGTH <= row && row < W_MAP_LENGTH * 2){
            BT(W_MAJOR) (W_mat, K, N, row, col) = -1;
        }else{
            BT(W_MAJOR) (W_mat, K, N, row, col) = 0;
        }
    }
}

/**
 * ここはroとcol orderで固定にする良さそう
 */
__global__ void tcMatMul(const signed char* const X,
                       int* const c){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, std::conditional_t<X_MAJOR == MAJOR_ROW, nvcuda::wmma::row_major, nvcuda::wmma::col_major>> X_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, std::conditional_t<W_MAJOR == MAJOR_ROW, nvcuda::wmma::row_major, nvcuda::wmma::col_major>> W_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> c_frag;

    nvcuda::wmma::fill_fragment(c_frag, 0);

#pragma unroll
    for(size_t k = 0; k < K; k += 16){
        if constexpr(X_MAJOR == MAJOR_ROW){
            nvcuda::wmma::load_matrix_sync(X_frag, X + (blockIdx.y * K * 16 + k), K);
        }else{
            nvcuda::wmma::load_matrix_sync(X_frag, X + (k * M + blockIdx.y * 16), M);
        }

        if constexpr(W_MAJOR == MAJOR_ROW){
            nvcuda::wmma::load_matrix_sync(W_frag, W_mat + ( k * N + blockIdx.x * 16), N);
        }else{
            nvcuda::wmma::load_matrix_sync(W_frag, W_mat + ( k + blockIdx.x * 16 * K), K);
        }
        nvcuda::wmma::mma_sync(c_frag, X_frag, W_frag, c_frag);
    }

    if constexpr(C_MAJOR == MAJOR_ROW){
        nvcuda::wmma::store_matrix_sync(c + (blockIdx.y * 16 * N + blockIdx.x * 16), c_frag, N, nvcuda::wmma::mem_row_major);
    }else{
        nvcuda::wmma::store_matrix_sync(c + (blockIdx.x * 16 * M + blockIdx.y * 16), c_frag, M, nvcuda::wmma::mem_col_major);
    }
}

__global__ void cuMatMul(const char* const X , int* const C){
    // CUDA内では2配列として使うことはできない。
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int start_col = (tid / M) * CALC_N_LENGTH;
    int row = tid % M;

#pragma unroll
    for(size_t col = start_col; col < start_col + CALC_N_LENGTH; col++){
        int accum = 0;
#pragma unroll
        for(size_t i = 0; i < W_MAP_LENGTH; i++){
            auto idx = BT(W_MAJOR) (W_map, W_MAP_LENGTH, N, i, col);
            accum += BT(X_MAJOR) (X, M, K, row, idx);
        }
        // indexを負の値にする方法では、なぜかパフォーマンスが劣化した
        // このため、別のmapとし作成することにより、パフォーマンスの劣化を抑える。
#pragma unroll
        for(size_t i = 0; i < W_MAP_LENGTH; i++){
            auto idx = BT(W_MAJOR) (W_map_negative, W_MAP_LENGTH, N, i, col);
            accum += -BT(X_MAJOR) (X, M, K, row, idx);
        }
        BT(C_MAJOR) (C, M, N, row, col) = accum;
    }
}

// assert uint8_t, col major, sm80
__device__ void make_map_a(unsigned tid, unsigned *i_map, unsigned *j_map){
    auto div_4 = tid / 4;
    auto mod_4 = tid % 4;

    for(unsigned i = 0; i < 4; i++){
        i_map[i] = div_4;
        j_map[i] = mod_4 * 4 + i;
    }
    for(unsigned i = 0; i < 4; i++){
        i_map[i + 4] = div_4 + 8;
        j_map[i + 4] = mod_4 * 4 + i;
    }
}

// assert uint8_t, col major, sm80
__device__ void make_map_b(unsigned tid, unsigned *i_map, unsigned *j_map){
    auto div_4 = tid / 4; // 0 0 0 0 1 1 1 1 2 2 2 2 3 3 3 3 4 4 4 4
    auto mod_4 = tid % 4; // 0 1 2 3 0 1 2 3 0 1 2 3

    for(unsigned i = 0; i < 4; i++){
        i_map[i] = mod_4 * 4 + i;
        j_map[i] = div_4;
    }
    for(unsigned i = 0; i < 4; i++){
        i_map[i + 4] = mod_4 * 4 + i;
        j_map[i + 4] = div_4 + 8;
    }
}


__global__ void newMatMul(const signed char* const X, int* const c){
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, signed char, std::conditional_t<X_MAJOR == MAJOR_ROW, nvcuda::wmma::row_major, nvcuda::wmma::col_major>> M_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, signed char, std::conditional_t<W_MAJOR == MAJOR_ROW, nvcuda::wmma::row_major, nvcuda::wmma::col_major>> I_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, int> c_frag;

    nvcuda::wmma::fill_fragment(c_frag, 0);
    nvcuda::wmma::fill_fragment(I_frag, 0);

    int land_id = mtk::wmma::detail::common::get_lane_id();

    unsigned b_i_map[8];
    unsigned b_j_map[8];
    make_map_b(land_id, b_i_map, b_j_map);

    for(unsigned f = 0; f < 8; f++){
        if(b_i_map[f] == b_j_map[f]){
            I_frag.x[f] = 1;
        }
    }

    __shared__  signed char M_tmp[16 * 16];

    for(unsigned k = 0; k < W_MAP_LENGTH; k++){
        int col_idx = BT(W_MAJOR) (W_map, W_MAP_LENGTH, N, k, blockIdx.x * 16 + (land_id % 16));

        for(unsigned i = 0; i < 8; i++){
            BT(X_MAJOR)(M_tmp, 16, 16, i + (land_id / 16 * 8), land_id % 16)
            = BT(X_MAJOR) (X, M, K, blockIdx.y * 16 + i + (land_id / 16 * 8), col_idx);
        }

        nvcuda::wmma::load_matrix_sync(M_frag, M_tmp, 16);
        nvcuda::wmma::mma_sync(c_frag, M_frag, I_frag, c_frag);

        col_idx = BT(W_MAJOR) (W_map_negative, W_MAP_LENGTH, N, k, blockIdx.x * 16 + (land_id % 16));

        for(unsigned i = 0; i < 8; i++){
            BT(X_MAJOR)(M_tmp, 16, 16, i + (land_id / 16 * 8), land_id % 16)
            = -BT(X_MAJOR) (X, M, K, blockIdx.y * 16 + i + (land_id / 16 * 8), col_idx);
        }

        nvcuda::wmma::load_matrix_sync(M_frag, M_tmp, 16);
        nvcuda::wmma::mma_sync(c_frag, M_frag, I_frag, c_frag);
    }


    if constexpr(C_MAJOR == MAJOR_ROW){
        nvcuda::wmma::store_matrix_sync(c + (blockIdx.y * 16 * N + blockIdx.x * 16), c_frag, N, nvcuda::wmma::mem_row_major);
    }else{
        nvcuda::wmma::store_matrix_sync(c + (blockIdx.x * 16 * M + blockIdx.y * 16), c_frag, M, nvcuda::wmma::mem_col_major);
    }
}


float measureKernel(std::function<void(void)> fn){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    fn();

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

void make_J(std::array<char, M * K> *X){
    X->fill(1);
}

int main(int argc, char** argv){

    static_assert(M % 16 == 0 && "mod 16 should be 0");
    static_assert(K % 16 == 0 && "mod 16 should be 0");
    static_assert(N % 16 == 0 && "mod 16 should be 0");
    static_assert(K < 65536 && "K should be fit in the maximum of short");

    char *X_d;
    hipMalloc((void**) &X_d, sizeof(char) * M * K);
    auto *X_ar = new std::array<char, M * K>(); make_J(X_ar);
    hipMemcpy(X_d, X_ar->data(), sizeof(char) * M * K, hipMemcpyHostToDevice);

    int *c_d; hipMalloc((void**)  &c_d, sizeof(int) * M * N ); hipMemset(c_d, 0, sizeof(int) * M * N);
    auto c_ar = new std::array<int, N * 1>(); // store only first row

    prepareW<<< N / 16, 16>>>();
    hipDeviceSynchronize();

    std::cout << "Start: " << "M=" << M << " K=" << K << " N=" << N << " ITER=" << ITER_NUM << " W_MAP_LENGTH=" << W_MAP_LENGTH << " CALC_N_LENGTH=" << CALC_N_LENGTH << std::endl;

    float ms = 0;

#ifdef RUN_TC
    ms = measureKernel([X_d, c_d](){
        for(size_t i = 0; i < ITER_NUM; i++){
            checkKernelErrors((tcMatMul<<< dim3(N / 16, M / 16) , 32>>>((signed char *) X_d, c_d)));
        }
    });
    std::cout << "TensorCore Time: " << ms / ((float) ITER_NUM) << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == 0 && "what");
    assert(c_ar->at(N / 2) == 0 && "what");
    assert(c_ar->at(N - 1) == 0 &&  "what");
#endif

#ifdef RUN_CUDA
    ms = measureKernel([X_d, c_d](){
        for(size_t i = 0; i < ITER_NUM; i++){
            checkKernelErrors((cuMatMul<<< N * M / (CALC_N_LENGTH * 32), 32 >>>(X_d, c_d)));
        }
    });
    std::cout << "CudaCore Time: " << ms / ((float) ITER_NUM) << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == 0 && "what");
    assert(c_ar->at(N / 2) == 0 && "what");
    assert(c_ar->at(N - 1) == 0 &&  "what");
#endif

#ifdef RUN_NEW
    ms = measureKernel([X_d, c_d](){
        for(size_t i = 0; i < ITER_NUM; i++){
            checkKernelErrors((newMatMul<<< dim3(N / 16, M / 16) , 32>>>((signed char *) X_d, c_d)));
        }
    });
    std::cout << "New Time: " << ms / ((float) ITER_NUM) << "ms" << std::endl;
    hipMemcpy(c_ar->data(), c_d, N * sizeof(int), hipMemcpyDeviceToHost);
    assert(c_ar->at(0) == 1 &&  "what");
    assert(c_ar->at(N / 2) == 0 &&  "what");
    assert(c_ar->at(N - 2) == 0 &&  "what");
#endif

    return 0;
}